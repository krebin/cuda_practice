
#include <hip/hip_runtime.h>
#include <iostream>

__global__
void vecAddKernel(float *A, float *B, float *C, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n)
    {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* h_A, float* h_B, float* h_C, int n)
{
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;
    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_B, size);
    hipMalloc((void **) &d_C, size);

    hipError_t err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__,__LINE__);
        exit(EXIT_FAILURE);
    }


    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main()
{
    float A[] = {1, 2, 3, 4};
    float B[] = {1, 2, 3, 4};
    float C[4] = {};
    vecAdd(A, B, C, 4);

    for(float i : A)
    {
        std::cout << i << " ";
    }
    std::cout << "\n";

    for(float i : B)
    {
        std::cout << i << " ";
    }
    std::cout << "\n";

    for(float i : C)
    {
        std::cout << i << " ";
    }
    std::cout << "\n";

}
